#include "hip/hip_runtime.h"
#include "texture.h"
#include "utils.h"
#include <iostream>
#include <hip/hip_fp16.h>


hipChannelFormatDesc get_channel_desc(int channels, int precision) {
    if (precision == PRECISION_FLOAT) {
        if (channels == 1) {
            return hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        }
        if (channels == 4) {
            return hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
        }
    }
    if (precision == PRECISION_HALF && channels == 4) {
        return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
    }
    std::cerr << "[TORCH RADON] ERROR unsupported number of channels and precision (channels:" << channels
              << ", precision: " << precision << ")" << std::endl;
    return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
}

Texture::Texture(DeviceSizeKey k) : key(k) {
    checkCudaErrors(hipSetDevice(this->key.device));

#ifdef VERBOSE
    std::cout << "[TORCH RADON] Allocating Texture " << this->key << std::endl;
#endif

    // Allocate a layered CUDA array
    hipChannelFormatDesc channelDesc = get_channel_desc(key.channels, key.precision);
    const hipExtent extent = make_hipExtent(k.width, k.height, k.batch / key.channels);
//    std::cout << k << std::endl;
    checkCudaErrors(hipMalloc3DArray(&array, &channelDesc, extent, hipArrayLayered));

    // Create resource descriptor
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    // Specify texture object parameters
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));

    // Create surface object
    checkCudaErrors(hipCreateSurfaceObject(&surface, &resDesc));
}

__global__ void
write_to_surface(const float *data, hipSurfaceObject_t surface, const int width, const int height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int b = 4 * blockIdx.z;

    if (x < width && y < height) {
        const int wh = width * height;
        const int offset = b * wh + y * width + x;

        float4 tmp;
        tmp.x = data[0 * wh + offset];
        tmp.y = data[1 * wh + offset];
        tmp.z = data[2 * wh + offset];
        tmp.w = data[3 * wh + offset];

        surf2DLayeredwrite<float4>(tmp, surface, x * sizeof(float4), y, blockIdx.z);
    }
}

void Texture::put(const float *data) {
    if (this->key.precision == PRECISION_HALF) {
        std::cerr << "[TORCH RADON] ERROR putting half precision data into a float texture" << std::endl;
    }

    checkCudaErrors(hipSetDevice(this->key.device));

    if (key.channels == 1) {
        // copy data into array
        hipMemcpy3DParms myparms = {0};
        myparms.srcPos = make_hipPos(0, 0, 0);
        myparms.dstPos = make_hipPos(0, 0, 0);
        myparms.srcPtr = make_hipPitchedPtr((void *) data, key.width * sizeof(float), this->key.width,
                                             this->key.height);
        myparms.dstArray = this->array;
        myparms.extent = make_hipExtent(this->key.width, this->key.height, this->key.batch);
        myparms.kind = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&myparms));
    } else {
        dim3 grid_dim(roundup_div(key.width, 16), roundup_div(key.height, 16), key.batch / 4);
        write_to_surface << < grid_dim, dim3(16, 16) >> > (data, this->surface, key.width, key.height);
    }

}

__global__ void
write_half_to_surface(const __half *data, hipSurfaceObject_t surface, const int width, const int height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int b = 4 * blockIdx.z;

    if (x < width && y < height) {
        const int wh = width * height;
        const int offset = b * wh + y * width + x;
        
        // Convert __half to float explicitly
        float tmp[4];
        for (int i = 0; i < 4; i++) {
            tmp[i] = __half2float(data[i * wh + offset]);
        }

        // Now create float2 from float values
        float2 float2_data = make_float2(tmp[0], tmp[1]);
        surf2DLayeredwrite<float2>(float2_data, surface, x * sizeof(float2), y, blockIdx.z);
    }
}
void Texture::put(const unsigned short *data) {
    if (this->key.precision == PRECISION_FLOAT) {
        std::cerr << "[TORCH RADON] ERROR putting single precision data into a half precision texture" << std::endl;
    }

    checkCudaErrors(hipSetDevice(this->key.device));

    dim3 grid_dim(roundup_div(key.width, 16), roundup_div(key.height, 16), key.batch / 4);
    write_half_to_surface << < grid_dim, dim3(16, 16) >> > ((__half *) data, this->surface, key.width, key.height);
}

bool Texture::matches(DeviceSizeKey &k) {
    return k == this->key;
}

Texture::~Texture() {
#ifdef VERBOSE
    std::cout << "[TORCH RADON] Freeing Texture " << this->key << std::endl;
#endif
    if (this->array != nullptr) {
        checkCudaErrors(hipSetDevice(this->key.device));
        checkCudaErrors(hipDestroyTextureObject(this->texture));
        checkCudaErrors(hipDestroySurfaceObject(this->surface));
        checkCudaErrors(hipFreeArray(this->array));
        this->array = nullptr;
    }
}