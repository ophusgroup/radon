#include <hip/hip_runtime.h>

#include <stdio.h>

// Simple CUDA kernel that adds two arrays
__global__ void add_arrays_kernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
        // Print hello from each thread (first few threads only to avoid spam)
        if (idx < 5) {
            printf("Hello from CUDA thread %d: %.2f + %.2f = %.2f\n", idx, a[idx], b[idx], c[idx]);
        }
    }
}

// Host function to launch the kernel
extern "C" {
    void launch_add_arrays(const float* a, const float* b, float* c, int n) {
        dim3 blockSize(256);
        dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

        add_arrays_kernel<<<gridSize, blockSize>>>(a, b, c, n);
        hipDeviceSynchronize();
    }
}
